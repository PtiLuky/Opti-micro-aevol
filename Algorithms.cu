#include "hip/hip_runtime.h"
#include "Algorithms.h"
#include "Algorithms.cuh"

#include "ExpManager.h"
#include "ThreefryGPU.h"
#include "GPUDna.cuh"

#include <cstdint>
#include <stdio.h>
#include <unistd.h>

#include <iostream>

#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

using namespace std;

#define DEBUG 1
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

#define SELECTION_SCOPE_X 3
#define SELECTION_SCOPE_Y 3
#define HALF_SCOPE_X 1
#define HALF_SCOPE_Y 1
#define NEIGHBORHOOD_SIZE 9


constexpr int32_t PROMOTER_ARRAY_SIZE = 10000;
uint32_t* gpu_sequences;
uint32_t* gpu_prev_sequences;
double* gpu_fitness;

/**
 * Function to transfer data from CPU to GPU
 *
 * @param exp_m
 * @param first_gen
 */
 void transfer_in(ExpManager* exp_m, bool first_gen) {
     exp_m->rng_->initDevice();

     // Malloc
     checkCuda(hipMalloc((void**) &gpu_counters,
                          exp_m->rng_->counters().size() *
                          sizeof(unsigned long long)));
    checkCuda(hipMalloc((void**) &gpu_prev_sequences,
                        exp_m->nb_indivs_ * 
                        exp_m->internal_organisms_[0]->dna_->seq_.size));
    
    checkCuda(hipMalloc((void**) &gpu_sequences, sizeof(gpu_prev_sequences)));
    
    checkCuda(hipMalloc((void**) &gpu_fitness, exp_m->nb_indivs_ * sizeof(double)));
    
    // Mem cpy
    checkCuda(hipMemcpy(gpu_counters, exp_m->rng_->counters().data(),
                        exp_m->rng_->counters().size() *
                        sizeof(unsigned long long), hipMemcpyHostToDevice));                    
    for(int i = 0; i < exp_m->nb_indivs_; ++i) {
        checkCuda(hipMemcpy(&gpu_prev_sequences[i], exp_m->internal_organisms_[i]->dna_->seq_.seq,
                            exp_m->internal_organisms_[0]->dna_->seq_.size, 
                            hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(&gpu_fitness[i], &(exp_m->internal_organisms_[i]->fitness),
                            sizeof(double), 
                            hipMemcpyHostToDevice));
        //printf("%f - ", exp_m->internal_organisms_[i]->fitness);
    }

     // TO COMPLETE
 }

 void transfer_out(ExpManager* exp_m) {
     // TODO
 }

 void clean(ExpManager* exp_m) {
    checkCuda(hipFree(gpu_counters));
    checkCuda(hipFree(gpu_sequences));
    checkCuda(hipFree(gpu_prev_sequences));
    // TO COMPLETE
}

__device__ int32_t Threefry::Device::roulette_random(double* probs, int32_t nb_elts)
{
    double pick_one = 0.0;

    while (pick_one == 0.0)
    {
        pick_one = randomDouble();
    }

    int32_t found_org = 0;

    pick_one -= probs[0];
    while (pick_one > 0)
    {
        assert(found_org<nb_elts-1);

        pick_one -= probs[++found_org];
    }
    return found_org;
}


__constant__ double cof[6] = {  76.18009172947146,
                                -86.50532032941677,
                                24.01409824083091,
                                -1.231739572450155,
                                0.1208650973866179e-2,
                                -0.5395239384953e-5 };



// Returns the value ln[gamma(X)] for X.
// The gamma function is defined by the integral  gamma(z) = int(0, +inf, t^(z-1).e^(-t)dt).
// When the argument z is an integer, the gamma function is just the familiar factorial
// function, but offset by one, n! = gamma(n + 1).
__device__ static double gammln(double X)
{
    double x, y, tmp, ser;

    y = x = X;
    tmp = x + 5.5;
    tmp -= (x+0.5) * log(tmp);
    ser = 1.000000000190015;

    for (int8_t j = 0 ; j <= 5 ; j++)
    {
        ser += cof[j] / ++y;
    }

    return -tmp + log(2.5066282746310005 * ser / x);
}


__device__ 
int32_t Threefry::Device::binomial_random(int32_t nb_drawings, double prob)
{
    int32_t nb_success;

    // The binomial distribution is invariant under changing
    // ProbSuccess to 1-ProbSuccess, if we also change the answer to
    // NbTrials minus itself; we ll remember to do this below.
    double p;
    if (prob <= 0.5) p = prob;
    else p = 1.0 - prob;

    // mean of the deviate to be produced
    double mean = nb_drawings * p;


    if (nb_drawings < 25)
        // Use the direct method while NbTrials is not too large.
        // This can require up to 25 calls to the uniform random.
    {
        nb_success = 0;
        for (int32_t j = 1 ; j <= nb_drawings ; j++)
        {
            if (randomDouble() < p) nb_success++;
        }
    }
    else if (mean < 1.0)
        // If fewer than one event is expected out of 25 or more trials,
        // then the distribution is quite accurately Poisson. Use direct Poisson method.
    {
        double g = exp(-mean);
        double t = 1.0;
        int32_t j;
        for (j = 0; j <= nb_drawings ; j++)
        {
            t = t * randomDouble();
            if (t < g) break;
        }

        if (j <= nb_drawings) nb_success = j;
        else nb_success = nb_drawings;
    }

    else
        // Use the rejection method.
    {
        double en     = nb_drawings;
        double oldg   = gammln(en + 1.0);
        double pc     = 1.0 - p;
        double plog   = log(p);
        double pclog  = log(pc);

        // rejection method with a Lorentzian comparison function.
        double sq = sqrt(2.0 * mean * pc);
        double angle, y, em, t;
        do
        {
            do
            {
                angle = M_PI * randomDouble();
                y = tan(angle);
                em = sq*y + mean;
            } while (em < 0.0 || em >= (en + 1.0)); // Reject.

            em = floor(em); // Trick for integer-valued distribution.
            t = 1.2 * sq * (1.0 + y*y)
                * exp(oldg - gammln(em + 1.0) - gammln(en - em + 1.0) + em * plog + (en - em) * pclog);

        } while (randomDouble() > t); // Reject. This happens about 1.5 times per deviate, on average.

        nb_success = (int32_t) rint(em);
    }


    // Undo the symmetry transformation.
    if (p != prob) nb_success = nb_drawings - nb_success;

    return nb_success;
}

__device__ static int mod(int a, int b)
{

    assert(b > 0);

    while (a < 0)  a += b;
    while (a >= b) a -= b;

    return a;
}

__global__ void selection(unsigned long long* gpu_counters, double* fitness, uint32_t* seqs, uint32_t* next,
    int nb_indiv, int grid_w, int grid_h, int size_seq){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int indiv_id = x * grid_h + y;

    if(x < grid_w && y < grid_h){
        double probs[NEIGHBORHOOD_SIZE];
        double tot_local_probs = 0;
        int count = 0;
        int cur_x;
        int cur_y;
        for(int i = - HALF_SCOPE_X; i  < SELECTION_SCOPE_X - HALF_SCOPE_X; ++i){
            for(int j = - HALF_SCOPE_Y; j < SELECTION_SCOPE_Y - HALF_SCOPE_Y; ++j){
                cur_x = mod(x+i, grid_w);
                cur_y = mod(y+j, grid_h);
    
                probs[count] = fitness[cur_x * grid_h + cur_y];
                tot_local_probs +=  probs[count];
    
                ++count;
            }
        }
        for(int i = 0 ; i < NEIGHBORHOOD_SIZE ; ++i) {
            probs[i] = probs[i]/tot_local_probs;
        }
    
        Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::REPROD,nb_indiv);
        int found_org = rng.roulette_random(probs, NEIGHBORHOOD_SIZE);
    
        int x_offset = (found_org / SELECTION_SCOPE_X) - HALF_SCOPE_X;
        int y_offset = (found_org % SELECTION_SCOPE_Y) - HALF_SCOPE_Y;
        cur_x = mod(x+x_offset, grid_w);
        cur_y = mod(y+y_offset, grid_h);
        int next_indiv_id = cur_x * grid_h + cur_y;
    
        for(int i = 0 ; i < size_seq ; ++i) {
            next[indiv_id * size_seq + i] = seqs[next_indiv_id * size_seq + i];
        }
    }
}
// seq_length is its true length in number of bool
__global__ void do_mutation(unsigned long long* gpu_counters, uint32_t* seqs,
    int nb_indiv, int grid_w, int grid_h, int seq_length, int size_seq, 
    double mutation_rate) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int indiv_id = x * grid_h + y;

    if(x < grid_w && y < grid_h){
        Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::MUTATION,nb_indiv);
        int nb_mut = rng.binomial_random(seq_length, mutation_rate);
        if(nb_mut > 0) {
            int nb_swi = nb_mut;
    
            while(nb_mut > 0) {
                int rand_val = rng.random(nb_mut);
                if(rand_val < nb_swi){
                    --nb_swi;
                    int pos = rng.random(seq_length);
                    dna_gpu_do_switch(seqs, indiv_id, size_seq, pos);
                }
                --nb_mut;
            }
        }
    }
}

/**
 * Run a step on the GPU
 * @param nb_indiv
 * @param seq_length number of bool in the seq of one indiv
 * @param w_max
 * @param selection_pressure
 * @param grid_width
 * @param grid_height
 * @param mutation_rate
 */
void run_a_step_on_GPU(int nb_indiv, int size_seq, int seq_length, double w_max, double selection_pressure, int grid_width, int grid_height, double mutation_rate) {
    dim3 DimGridOrganism(ceil(grid_width/16.),ceil(grid_height/16.),1);
    dim3 DimBlockOrganism(16,16,1);

    selection<<<DimGridOrganism, DimBlockOrganism>>>(gpu_counters, gpu_fitness, gpu_prev_sequences, gpu_sequences, 
        nb_indiv, grid_width, grid_height, size_seq);
    checkCuda(hipGetLastError());

    do_mutation<<<DimGridOrganism, DimBlockOrganism>>>(gpu_counters, gpu_sequences, 
        nb_indiv, grid_width, grid_height, seq_length, size_seq, mutation_rate);
    checkCuda(hipGetLastError());

    apply_next_gen();
}

/**
 * Reallocate some data structures if needed
 * @param nb_indiv
 */
void apply_next_gen() {
    uint32_t* temp = gpu_prev_sequences;
    gpu_prev_sequences = gpu_sequences;
    gpu_sequences = temp;
}

/**
PRNG usage:
 * For selection
        Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::REPROD,nb_indiv);
        int found_org = rng.roulette_random(probs, NEIGHBORHOOD_SIZE);
 * For mutation:
      Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::MUTATION,nb_indiv);
      rng.binomial_random(prev_gen_size, mutation_r);
      rng.random( number );
 **/